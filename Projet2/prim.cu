#include "hip/hip_runtime.h"
#ifndef PRIME_CPP
#define PRIME_CPP
#include "prim.h"
using namespace std;
float pi = std::acos(-1);

#define deg_to_rad_fac (M_PI / 180.0)
static inline int fls(int x)
{
int position;
int i;
if(0 != x)
{
for (i = (x >> 1), position = 0; i != 0; ++position)
           i >>= 1;
}
else
{
        position = -1;
} 
    return position+1;
}
static inline unsigned int roundup_pow_of_two(unsigned int x)
{
    return 1UL << fls(x - 1);
}

__device__ __host__ inline double sin_deg(double a){
        return (sin(a * deg_to_rad_fac));
}

__device__ __host__ inline double cos_deg(double a){
        return (cos(a * deg_to_rad_fac));
}


__device__ inline float getDistance(double sin_lat1,double cos_lat1, double lon1, double sin_lat2, double cos_lat2, double lon2){
  return  float(R_EARTH * acos(fmin(sin_lat1 * sin_lat2 
             + cos_deg(lon1 - lon2) * cos_lat1* cos_lat2,1.0)));
}

__global__ void compute_sincoslat_functor(float * lat, double * sin_lat, double* cos_lat,const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < maxN){
    double temp = lat[i] * deg_to_rad_fac;
    sincos(temp, &(sin_lat[i]), &(cos_lat[i]) );
  }
};




__global__ void init_dist_functor(double * sin_lat, double* cos_lat, float* lon, float * min_dist,int* parent, float* distance_total_dev, const int compare_index, const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < maxN){
    if(i == compare_index)  min_dist[i] = FLT_MAX;
    else min_dist[i] = getDistance(sin_lat[i], cos_lat[i], lon[i], sin_lat[compare_index], cos_lat[compare_index],lon[compare_index]);
    
    parent[i] = compare_index;

  }
  if(i == 0){
    distance_total_dev[0] = 0.0;
  }
};

__global__ void  update_min_dist_functor(float* min_dist, double* sin_lat, double* cos_lat, float*  lon, int* parent,int* new_in_index, const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < maxN)
  {  
    if(min_dist[i] >= FLT_MAX){
      return;
    }
    else{
      float temp = getDistance(sin_lat[i], cos_lat[i], lon[i],sin_lat[(*new_in_index)],cos_lat[(*new_in_index)], lon[(*new_in_index)]);
      if(temp < min_dist[i]){
        
        min_dist[i] = temp;
        
        parent[i] = (*new_in_index);
      }
    }
  }
};



__global__ void reduce_find_min_block(float *min_dist, int * min_min_index_block, float*min_min_dist_block, int nbVilles) 
{
 
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x *blockDim.x + threadIdx.x;
  int nTotalThreads = blockDim.x;
  // each thread loads one element from global to shared mem
  //unsigned int tid = threadIdx.x;
  __shared__ float min[256];
  __shared__ int min_index[256];
  if(i<nbVilles){
    min[tid] = min_dist[i];
    min_index[tid] = i;
  }
  else{
    min[tid] = FLT_MAX;
    min_index[tid] = -1;
  }
  __syncthreads();


  while(nTotalThreads > 1)
  {
    int quaterPoint = (nTotalThreads >> 2); // divide by two
    if (tid < quaterPoint)
    {
      int thread2 = tid + quaterPoint;
      if(min[thread2] < min[tid] ){
        min[tid] = min[thread2];
        min_index[tid] = min_index[thread2];
      }
      thread2 = tid + 2* quaterPoint;
      if(min[thread2] < min[tid] ){
        min[tid] = min[thread2];
        min_index[tid] = min_index[thread2];
      }
      thread2 = tid + 3* quaterPoint;
      if(min[thread2] < min[tid] ){
        min[tid] = min[thread2];
        min_index[tid] = min_index[thread2];
      }
    }
    __syncthreads();
    nTotalThreads = quaterPoint;
  }

  if(tid == 0){
  
    min_min_index_block[blockIdx.x] = min_index[0];
    min_min_dist_block[blockIdx.x] = min[0];
    //min_dist[(*min_min_index)] = -1.0;
    //*distance_total_dev += (*min_min_dist);
  }
  
}
__global__ void reduce_find_min(float* min_dist,int * min_min_index_block, float*min_min_dist_block, float* distance_total_dev, int* min_min_index, float* min_min_dist, int block_size) 
{
  __shared__ float min[512];
  __shared__ int min_index[512];

  unsigned int i = threadIdx.x;
  int nTotalThreads = blockDim.x;
  // each thread loads one element from global to shared mem
  //unsigned int tid = threadIdx.x;
  if(i < block_size){
    min[i] = min_min_dist_block[i];
    min_index[i] = min_min_index_block[i];
  }
  else{
    min[i] = FLT_MAX;
    min_index[i] = -1;
  }
    __syncthreads();


    while(nTotalThreads > 1)
    {
      int quaterPoint = (nTotalThreads >> 2); // divide by two
      if (i < quaterPoint)
      {
        int thread2 = i + quaterPoint;
        if(min[thread2] < min[i]){
          min[i] = min[thread2];
          min_index[i] = min_index[thread2];
        }
        thread2 = i + 2 * quaterPoint;
        if(min[thread2] < min[i]){
          min[i] = min[thread2];
          min_index[i] = min_index[thread2];
        }
        thread2 = i + 3 * quaterPoint;
        if(min[thread2] < min[i]){
          min[i] = min[thread2];
          min_index[i] = min_index[thread2];
        }
      }
      __syncthreads();
      nTotalThreads = quaterPoint;
    }
    if(i == 0){
    
      *min_min_index = min_index[0];
      *min_min_dist = min[0];
      min_dist[(*min_min_index)] = FLT_MAX;
      *distance_total_dev += (*min_min_dist);
    }
  
}
void prim(float *lat_host, float* lon_host, int* parent_host,
          const int nbVilles, float *distance_total)
{
  //int threadsPerBlock = 256;
  //int blocksPerGrid =(nbVilles + threadsPerBlock - 1) / threadsPerBlock;

  dim3 threadsPerBlock(256); 
  const int block_size_ =  (nbVilles + threadsPerBlock.x - 1) / threadsPerBlock.x;
  dim3 blocksPerGrid(block_size_);  
  
  // Allocate the device input vector 
  float *lat_dev = NULL;
  float *lon_dev = NULL;
  int *parent_dev = NULL;
  float *min_dist_dev = NULL;
  float  *distance_total_dev = NULL;
  int* min_min_index_dev = NULL;
  float* min_min_dist_dev = NULL;
  int* min_min_index_dev_block = NULL;
  float* min_min_dist_dev_block = NULL;
  double* sin_lat_dev = NULL;
  double* cos_lat_dev = NULL;


  const int memory_size_float =  nbVilles * sizeof(float);
  const int memory_size_int =  nbVilles * sizeof(int);
  hipMalloc((void **)&distance_total_dev, sizeof(float));
  hipMalloc((void **)&min_min_index_dev, sizeof(int));
  hipMalloc((void **)&min_min_dist_dev, sizeof(float));
  hipMalloc((void **)&min_min_index_dev_block, block_size_*sizeof(int));
  hipMalloc((void **)&min_min_dist_dev_block, block_size_*sizeof(float));
  hipMalloc((void **)&sin_lat_dev, nbVilles*sizeof(double));
  hipMalloc((void **)&cos_lat_dev, nbVilles*sizeof(double));


  hipError_t err_lat = hipMalloc((void **)&lat_dev, memory_size_float);
  hipError_t err_lon = hipMalloc((void **)&lon_dev, memory_size_float);
  hipError_t err_min_dist = hipMalloc((void **)&min_dist_dev, memory_size_float);
  hipError_t err_parent = hipMalloc((void **)&parent_dev, memory_size_int);

  //float *debug_min_dist_dev = NULL;
  //hipMalloc((void **)&debug_min_dist_dev, nbVilles*sizeof(float));
  //float *debug_min_dist_host = (float*) malloc(nbVilles*sizeof(float));
  if (err_lat != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector lat_dev(error code %s)!\n", hipGetErrorString(err_lat));
      exit(EXIT_FAILURE);
  }
  if (err_lon != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector lon_dev(error code %s)!\n", hipGetErrorString(err_lon));
      exit(EXIT_FAILURE);
  }
  if (err_parent != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector parent_dev(error code %s)!\n", hipGetErrorString(err_parent));
      exit(EXIT_FAILURE);
  }
  if (err_min_dist != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector min_dist_dev(error code %s)!\n", hipGetErrorString(err_min_dist));
      exit(EXIT_FAILURE);
  }

  // copy to GPU memory
  err_lat = hipMemcpy(lat_dev, lat_host, memory_size_float, hipMemcpyHostToDevice);
  err_lon = hipMemcpy(lon_dev, lon_host, memory_size_float, hipMemcpyHostToDevice);
  err_parent = hipMemcpy(parent_dev, parent_host, memory_size_int, hipMemcpyHostToDevice);
  if (err_lat != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector lat from host to device (error code %s)!\n", hipGetErrorString(err_lat));
      exit(EXIT_FAILURE);
  }
  if (err_lon != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector lon from host to device (error code %s)!\n", hipGetErrorString(err_lon));
      exit(EXIT_FAILURE);
  }
  if (err_parent != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector parent from host to device (error code %s)!\n", hipGetErrorString(err_parent));
      exit(EXIT_FAILURE);
  }
  


  // init Prim
  /* 
	for(i = 1; i < nbVilles; i++)
	{
		min_dist[i] = distance[0][i];
		parent[i] = 0;
	}
  */
  compute_sincoslat_functor<<<blocksPerGrid, threadsPerBlock>>>(lat_dev, sin_lat_dev, cos_lat_dev,nbVilles);
  init_dist_functor<<<blocksPerGrid, threadsPerBlock>>>(sin_lat_dev,cos_lat_dev, lon_dev, min_dist_dev, parent_dev,distance_total_dev, 0, nbVilles);

  hipDeviceSynchronize();
  ////cout<< "init: ==================="<<endl;
  //float * min_dist_host = (float*)malloc(nbVilles * sizeof(float));
  //hipMemcpy(min_dist_host, min_dist_dev, memory_size_float, hipMemcpyDeviceToHost);
  //hipMemcpy(parent_host, parent_dev,memory_size_int, hipMemcpyDeviceToHost);
  //hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
  //debug_print(min_dist_host, parent_host,*distance_total, nbVilles);


	hipError_t err = hipGetLastError();
  if (err != hipSuccess )
  {
      fprintf(stderr, "Failed to run kernel init_dist_functor(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  


  //iteration of Prime
  int k;

  //for debug
  //float * min_min_dist_host_block = (float*)malloc(block_size_ * sizeof(float));
  //int * min_min_index_host_block = (int*)malloc(block_size_ * sizeof(int));
  //float * min_dist_host = (float*)malloc(nbVilles * sizeof(float));
  //hipMemcpy(min_dist_host, min_dist_dev, nbVilles * sizeof(float), hipMemcpyDeviceToHost);
  //validation
  //float* debug_min_dist_validation_host = (float*)malloc(memory_size_float);
  //for(int i = 0 ; i<nbVilles; i++)
  // debug_min_dist_validation_host[i] = R_EARTH * std::acos( std::min(sin_deg(lat_host[i]) * sin_deg(lat_host[0])  + cos_deg(lon_host[i]-lon_host[0]) * cos_deg(lat_host[i])* cos_deg(lat_host[0]), 1.0));

  //debug_print(debug_min_dist_validation_host, min_dist_host, nbVilles);
  //cout << "-------------------"<<endl;
  

	for(k = 0; k < nbVilles -1 ; k++)
	{
    /*
		// find the minimal min_dist outstide of S
		min_min_dist = FLT_MAX;

		for(i = 0; i < nbVilles; i++)
		{
      __assume_aligned(min_dist, VEC_ALIGN);
				if(min_dist[i] >0 && min_min_dist > min_dist[i])
				{
					min_min_dist = min_dist[i];
					min_min_dist_index = i;
				}
		}
  */
    //hipMemcpy(min_dist_host, min_dist_dev, nbVilles * sizeof(float), hipMemcpyDeviceToHost);
    
    //debug_print(min_dist_host, nbVilles );
    //cout<< "<<<<<<<<<<" <<endl;
    reduce_find_min_block<<<blocksPerGrid,threadsPerBlock >>>(min_dist_dev, min_min_index_dev_block, min_min_dist_dev_block, nbVilles);
    hipDeviceSynchronize();
    if (err != hipSuccess )
    {
        fprintf(stderr, "Failed to run kernel reduce_find_min_block(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    
    //hipMemcpy(min_min_index_host_block, min_min_index_dev_block, block_size_ * sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(min_min_dist_host_block, min_min_dist_dev_block, block_size_ * sizeof(float), hipMemcpyDeviceToHost);
    
    //debug_print(min_min_index_host_block,min_min_dist_host_block, block_size_ );
    //cout << "============"<<endl;
    //cout << "std min dist:" << *(min_element(min_dist_host, min_dist_host+nbVilles))<<endl;
    //cout << "block_size: "<<block_size_ << " nbVilles: "<< nbVilles <<endl;
    reduce_find_min<<<1,  roundup_pow_of_two(block_size_)>>>(min_dist_dev, min_min_index_dev_block, min_min_dist_dev_block,distance_total_dev, min_min_index_dev,min_min_dist_dev,block_size_);
    err = hipGetLastError();
    if (err != hipSuccess )
    {
        fprintf(stderr, "Failed to run kernel reduce_find_min(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //float min_min_dist_host;
    //int min_min_index_host;
    //hipMemcpy(min_dist_host, min_dist_dev, memory_size_float, hipMemcpyDeviceToHost);
    //hipMemcpy(parent_host, parent_dev,memory_size_int, hipMemcpyDeviceToHost);
    //hipMemcpy(&min_min_index_host, min_min_index_dev,sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(&min_min_dist_host, min_min_dist_dev, sizeof(float), hipMemcpyDeviceToHost);
    //debug_print(min_dist_host, parent_host,*distance_total, min_min_index_host, k, nbVilles);
    //cout<<"total: "<< nbVilles << " min_min_index: "<<min_min_index_host<< " min_min_dist: "<< min_min_dist_host<< " dist_toal: "<<*distance_total<<endl;
    
    //debug_print(min_dist_host, parent_host,*distance_total, min_min_index_host, k, nbVilles);

    //float min_min_dist_host;
    //float min_min_index_host;
    //hipMemcpy(&min_min_index_host, min_min_index_dev, sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(&min_min_dist_host, min_min_dist_dev, sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
    ////cout << "min_dist: "<< min_min_dist_host << "index: "<< min_min_index_host << " distance_total: "<< *distance_total<<endl;

    update_min_dist_functor<<<blocksPerGrid, threadsPerBlock>>>(min_dist_dev, sin_lat_dev, cos_lat_dev, lon_dev, parent_dev ,min_min_index_dev, nbVilles);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess )
    {
        fprintf(stderr, "Failed to run kernel update_min_dist_functor(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //hipMemcpy(debug_min_dist_host, debug_min_dist_dev, memory_size_float, hipMemcpyDeviceToHost);
    //hipMemcpy(min_dist_host, min_dist_dev, memory_size_float, hipMemcpyDeviceToHost);
    //hipMemcpy(parent_host, parent_dev,memory_size_int, hipMemcpyDeviceToHost);
    //debug_print(debug_min_dist_host, min_dist_host, nbVilles);

        #ifdef SHOW_EVERY_STEPS
        // check and create folder "steps"
        int iRet = access("./steps",0);
        if (iRet != 0)
                iRet = mkdir("./steps",S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);

        // create files steps files to store data
        string save_name_base = "./steps/resuGraphe_";
        string save_name = save_name_base + to_string(k)+".dat";
        ////cout<<"steps: "<< 0<<endl;
        ofstream fileOut(save_name);
        for(int i =0; i < nbVilles; i++)
        {
        fileOut << parent_host[i] << " "<< i <<" "<<min_dist_host[i] << " " << int(min_dist_host[i] >= (FLT_MAX-10)) <<"\n";
        }
        fileOut.close();
        #endif // SHOW_EVERY_STEPS




  }
  err_parent = hipMemcpy(parent_host, parent_dev, memory_size_int, hipMemcpyDeviceToHost);
  hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(distance_total_dev);
  hipFree(min_min_index_dev);
  hipFree(min_min_dist_dev);
  hipFree(lat_dev);
  hipFree(lon_dev);
  hipFree(min_dist_dev);
  hipFree(parent_dev);
  //hipFree(debug_min_dist_dev);
  //free(min_min_dist_host_block);
  //free(min_min_index_host_block);
  //free(min_dist_host);
  //free(debug_min_dist_host);
  //free(debug_min_dist_validation_host);
}


#endif
