#include "hip/hip_runtime.h"
#ifndef PRIME_CPP
#define PRIME_CPP
#include "prim.h"
using namespace std;
double pi = std::acos(-1);

#define deg_to_rad_fac (M_PI / 180.0)
static inline int fls(int x)
{
int position;
int i;
if(0 != x)
{
for (i = (x >> 1), position = 0; i != 0; ++position)
           i >>= 1;
}
else
{
        position = -1;
} 
    return position+1;
}
static inline unsigned int roundup_pow_of_two(unsigned int x)
{
    return 1UL << fls(x - 1);
}

__device__ inline double sin_deg(double a){
        return (sin(a * deg_to_rad_fac));
}

__device__ inline double cos_deg(double a){
        return (cos(a * deg_to_rad_fac));
}


__device__ inline float getDistance(double lat1, double lon1, double lat2, double lon2){
  return  float(R_EARTH * acos(min(sin(lat1) * sin(lat2) 
             + cos_deg(lon1 - lon2) * cos(lat1)* cos(lat2),1.0)));
}
__global__ void compute_dist_functor(float * lat, float* lon, float * min_dist,const int compare_index, const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < maxN){
    min_dist[i] = getDistance(lat[i], lon[i], lat[compare_index],lon[compare_index]);
  }
};

__global__ void init_dist_functor(float * lat, float* lon, float * min_dist,int* parent, float* distance_total_dev, const int compare_index, const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < maxN){
    if(i == compare_index)  min_dist[i] = -1.0;
    else min_dist[i] = getDistance(lat[i], lon[i], lat[compare_index],lon[compare_index]);
    
    parent[i] = compare_index;

  }
  if(i == 0){
    distance_total_dev[0] = 0.0;
  }
};

__global__ void  update_min_dist_functor(float* min_dist, float* lat, float*  lon, int* parent,int* new_in_index, const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < maxN)
  {  
    if(min_dist[i] < 0)
      return;
    else{
      float temp = getDistance(lat[i], lon[i],lat[(*new_in_index)], lon[(*new_in_index)]);
      if(temp < min_dist[i]){
        parent[i] = (*new_in_index);
      }
    }
  }
};


__global__ void  set_all_parent_functor(int * parent,int parent_default, const int maxN)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < maxN){  
      parent[i] = parent_default;    
  }
};


__global__ void reduce_find_min_block(float *min_dist, int * min_min_index_block, float*min_min_dist_block, int nbVilles) 
{
  __shared__ float min[256];
  __shared__ int min_index[256];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x *blockDim.x + threadIdx.x;
  int nTotalThreads = blockDim.x;
  // each thread loads one element from global to shared mem
  //unsigned int tid = threadIdx.x;
  if(i < nbVilles){
    min[tid] = min_dist[i];
    min_index[tid] = i,
    __syncthreads();


    while(nTotalThreads > 1)
    {
      int halfPoint = (nTotalThreads >> 1); // divide by two
      if (tid < halfPoint)
      {
        int thread2 = tid + halfPoint;
        if(min[thread2] >= 0 && (min[thread2] < min[tid] || min[tid] < 0)){
          min[tid] = min[thread2];
          min_index[tid] = thread2;
        }
      }
      __syncthreads();
      nTotalThreads = halfPoint;
    }
    if(tid == 0){
    
      min_min_index_block[blockIdx.x] = min_index[0];
      min_min_dist_block[blockIdx.x] = min[0];
      //min_dist[(*min_min_index)] = -1.0;
      //*distance_total_dev += (*min_min_dist);
    }
  }
}
__global__ void reduce_find_min(float* min_dist,int * min_min_index_block, float*min_min_dist_block, float* distance_total_dev, int* min_min_index, float* min_min_dist, int block_size) 
{
  __shared__ float min[512];
  __shared__ int min_index[512];

  unsigned int i = threadIdx.x;
  int nTotalThreads = blockDim.x;
  // each thread loads one element from global to shared mem
  //unsigned int tid = threadIdx.x;
  if(i < block_size){
    min[i] = min_min_dist_block[i];
    min_index[i] = min_min_index_block[i],
    __syncthreads();


    while(nTotalThreads > 1)
    {
      int halfPoint = (nTotalThreads >> 1); // divide by two
      if (i < halfPoint)
      {
        int thread2 = i + halfPoint;
        if(min[thread2] >= 0 && (min[thread2] < min[i] || min[i] < 0)){
          min[i] = min[thread2];
          min_index[i] = thread2;
        }
      }
      __syncthreads();
      nTotalThreads = halfPoint;
    }
    if(i == 0){
    
      *min_min_index = min_index[0];
      *min_min_dist = min[0];
      min_dist[(*min_min_index)] = -1.0;
      *distance_total_dev += (*min_min_dist);
    }
  }
}


void prim(float *lat_host, float* lon_host, int* parent_host,
          const int nbVilles, float *distance_total)
{
  //int threadsPerBlock = 256;
  //int blocksPerGrid =(nbVilles + threadsPerBlock - 1) / threadsPerBlock;

  dim3 threadsPerBlock(128); 
  const int block_size =  (nbVilles + threadsPerBlock.x - 1) / threadsPerBlock.x;
  dim3 blocksPerGrid(block_size);  

  // Allocate the device input vector 
  float *lat_dev = NULL;
  float *lon_dev = NULL;
  int *parent_dev = NULL;
  float *min_dist_dev = NULL;
  float  *distance_total_dev = NULL;
  int* min_min_index_dev = NULL;
  float* min_min_dist_dev = NULL;
  int* min_min_index_dev_block = NULL;
  float* min_min_dist_dev_block = NULL;
  const int memory_size_float =  nbVilles * sizeof(float);
  const int memory_size_int =  nbVilles * sizeof(float);
  hipMalloc((void **)&distance_total_dev, sizeof(float));
  hipMalloc((void **)&min_min_index_dev, sizeof(int));
  hipMalloc((void **)&min_min_dist_dev, sizeof(float));
  hipMalloc((void **)&min_min_index_dev_block, block_size*sizeof(int));
  hipMalloc((void **)&min_min_dist_dev_block, block_size*sizeof(float));


  hipError_t err_lat = hipMalloc((void **)&lat_dev, memory_size_float);
  hipError_t err_lon = hipMalloc((void **)&lon_dev, memory_size_float);
  hipError_t err_min_dist = hipMalloc((void **)&min_dist_dev, memory_size_float);
  hipError_t err_parent = hipMalloc((void **)&parent_dev, memory_size_int);
  if (err_lat != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector lat_dev(error code %s)!\n", hipGetErrorString(err_lat));
      exit(EXIT_FAILURE);
  }
  if (err_lon != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector lon_dev(error code %s)!\n", hipGetErrorString(err_lon));
      exit(EXIT_FAILURE);
  }
  if (err_parent != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector parent_dev(error code %s)!\n", hipGetErrorString(err_parent));
      exit(EXIT_FAILURE);
  }
  if (err_min_dist != hipSuccess )
  {
      fprintf(stderr, "Failed to allocate device vector min_dist_dev(error code %s)!\n", hipGetErrorString(err_min_dist));
      exit(EXIT_FAILURE);
  }

  // copy to GPU memory
  err_lat = hipMemcpy(lat_dev, lat_host, memory_size_float, hipMemcpyHostToDevice);
  err_lon = hipMemcpy(lon_dev, lon_host, memory_size_float, hipMemcpyHostToDevice);
  err_parent = hipMemcpy(parent_dev, parent_host, memory_size_int, hipMemcpyHostToDevice);
  if (err_lat != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector lat from host to device (error code %s)!\n", hipGetErrorString(err_lat));
      exit(EXIT_FAILURE);
  }
  if (err_lon != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector lon from host to device (error code %s)!\n", hipGetErrorString(err_lon));
      exit(EXIT_FAILURE);
  }
  if (err_parent != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector parent from host to device (error code %s)!\n", hipGetErrorString(err_parent));
      exit(EXIT_FAILURE);
  }
  


  // init Prim
  /* 
	for(i = 1; i < nbVilles; i++)
	{
		min_dist[i] = distance[0][i];
		parent[i] = 0;
	}
  */
  init_dist_functor<<<blocksPerGrid, threadsPerBlock>>>(lat_dev, lon_dev, min_dist_dev, parent_dev,distance_total_dev, 0, nbVilles);

  hipDeviceSynchronize();
  //cout<< "init: ==================="<<endl;
  //float * min_dist_host = (float*)malloc(nbVilles * sizeof(float));
  //hipMemcpy(min_dist_host, min_dist_dev, memory_size_float, hipMemcpyDeviceToHost);
  //hipMemcpy(parent_host, parent_dev,memory_size_int, hipMemcpyDeviceToHost);
  //hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
  //debug_print(min_dist_host, parent_host,*distance_total, nbVilles);


	hipError_t err = hipGetLastError();
  if (err != hipSuccess )
  {
      fprintf(stderr, "Failed to run kernel init_dist_functor(error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  
  //iteration of Prime
  int k;
	for(k = 0; k < nbVilles -1 ; k++)
	{
    /*
		// find the minimal min_dist outstide of S
		min_min_dist = FLT_MAX;

		for(i = 0; i < nbVilles; i++)
		{
      __assume_aligned(min_dist, VEC_ALIGN);
				if(min_dist[i] >0 && min_min_dist > min_dist[i])
				{
					min_min_dist = min_dist[i];
					min_min_dist_index = i;
				}
		}
  */
//reduce_find_min_block(float *min_dist, int * min_min_index_block, float*min_min_dist_block, int nbVilles, int array_size) 
//reduce_find_min(float* min_dist,int * min_min_index_block, float*min_min_dist_block, float* distance_total_dev, int* min_min_index, float* min_min_dist) 
    //cout <<"roudup : "<< roundup_pow_of_two(nbVilles) << endl;
    reduce_find_min_block<<<blocksPerGrid, threadsPerBlock>>>(min_dist_dev, min_min_index_dev_block, min_min_dist_dev_block, nbVilles);
    hipDeviceSynchronize();
    if (err != hipSuccess )
    {
        fprintf(stderr, "Failed to run kernel reduce_find_min_block(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    reduce_find_min<<<1,  blocksPerGrid>>>(min_dist_dev, min_min_index_dev_block, min_min_dist_dev_block,distance_total_dev, min_min_index_dev,min_min_dist_dev,block_size);
    err = hipGetLastError();
    if (err != hipSuccess )
    {
        fprintf(stderr, "Failed to run kernel reduce_find_min(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
  

    //float min_min_dist_host;
    //float min_min_index_host;
    //hipMemcpy(&min_min_index_host, min_min_index_dev, sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(&min_min_dist_host, min_min_dist_dev, sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
    //cout << "min_dist: "<< min_min_dist_host << "index: "<< min_min_index_host << " distance_total: "<< *distance_total<<endl;

    //update the min_dist
    /*
    int dist_temp;

    float* distance_i_ptr =  distance[min_min_dist_index];
		for(j = 0; j < nbVilles; j++)
		{
      __assume_aligned(distance_i_ptr, VEC_ALIGN);
      dist_temp = distance_i_ptr[j];

			if(min_dist[j] > dist_temp)
			{
				min_dist[j] = dist_temp;
				parent[j] = min_min_dist_index;
			}
		}
    */
    update_min_dist_functor<<<blocksPerGrid, threadsPerBlock>>>(min_dist_dev, lat_dev, lon_dev, parent_dev ,min_min_index_dev, nbVilles);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess )
    {
        fprintf(stderr, "Failed to run kernel update_min_dist_functor(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
  }
  err_parent = hipMemcpy(parent_host, parent_dev, memory_size_int, hipMemcpyDeviceToHost);
  hipMemcpy(distance_total, distance_total_dev, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(distance_total_dev);
  hipFree(min_min_index_dev);
  hipFree(min_min_dist_dev);
  hipFree(lat_dev);
  hipFree(lon_dev);
  hipFree(min_dist_dev);
  hipFree(parent_dev);
  //free(min_dist_host);
}


#endif
