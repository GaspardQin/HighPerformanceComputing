#include "hip/hip_runtime.h"
#ifndef PRIME_CPP
#define PRIME_CPP
#include "prim.h"
using namespace std;
double pi = std::acos(-1);

#define deg_to_rad_fac (M_PI / 180.0)

__host__ __device__
inline thrust::complex<double> sin_deg(thrust::complex<double> a){
        return (thrust::sin(a * deg_to_rad_fac));
}
__host__ __device__
inline thrust::complex<double> cos_deg(thrust::complex<double> a){
        return (thrust::cos(a * deg_to_rad_fac));
}


__host__ __device__
inline double getDistance(double lat1, double lon1, double lat2, double lon2){
  thrust::complex<double> complex_lat1 = thrust::complex<double>(lat1);
  thrust::complex<double> complex_lat2 = thrust::complex<double>(lat2);
  thrust::complex<double> complex_lon1 = thrust::complex<double>(lon1);
  thrust::complex<double> complex_lon2 = thrust::complex<double>(lon2);

  return  R_EARTH * thrust::norm(thrust::acos( thrust::sin(complex_lat1) * thrust::sin(complex_lat2) 
             + cos_deg(complex_lon1 - complex_lon2) * thrust::cos(complex_lat1)* thrust::cos(complex_lat2)));
}


struct compute_dist_functor
{
    POS * pos_ptr;
    int compare_index;
    compute_dist_functor(POS * pos_ptr_, int compare_index_): pos_ptr(pos_ptr_), compare_index(compare_index_){};
    __host__ __device__
        void operator()(const int& a) const{ 
          (pos_ptr+a)->min_dist = getDistance((pos_ptr+a)->lat, (pos_ptr+a)->lon, (pos_ptr+compare_index)->lat,(pos_ptr+compare_index)->lon);
        }
};


struct update_min_dist_functor
{
  thrust::device_vector<POS>::iterator pos_ptr;
  int new_in_index;
  double temp;
  POS pos_a;
  POS pos_new_index;
  update_min_dist_functor(thrust::device_vector<POS>::iterator pos_ptr_, int new_in_index_): pos_ptr(pos_ptr_), new_in_index(new_in_index_){
    pos_new_index = *(pos_ptr+new_in_index);
  };
  
  __host__ __device__
  void operator()(const int& a)
  {
    pos_a = *(pos_ptr+a);
    if(pos_a.is_in == true)
      return;
    else{
      temp = getDistance(pos_a.lat, pos_a.lon, pos_new_index.lat,pos_new_index.lon);
      if(temp < pos_a.min_dist){
        pos_a.parent = new_in_index;
        *(pos_ptr+a) = pos_a;
      }
    }
  }
};


struct set_all_parent_functor
{
    int parent_default;
    
    set_all_parent_functor(int a):parent_default(a){};
    
    __host__ __device__
    void operator()(POS& A){ 
      A.parent = parent_default;    
    }
};
struct compute_min_dist_functor{
    thrust::device_vector<POS>::iterator pos_ptr;
    int compare_index;
    POS pos_compare_copy;
    POS pos_a_copy;
    compute_min_dist_functor(thrust::device_vector<POS>::iterator pos_ptr_ , int compare_index_){
      pos_ptr = pos_ptr_;
      compare_index = compare_index_;
      pos_compare_copy = *(pos_ptr+compare_index);
    }
    
    __host__ __device__
        void operator()(const int& a){
            pos_a_copy = *(pos_ptr + a); 
            pos_a_copy.min_dist = getDistance(pos_a_copy.lat, pos_a_copy.lon, pos_compare_copy.lat,pos_compare_copy.lon);
            *(pos_ptr + a) = pos_a_copy;
        }
};


struct compare_min_dist_functor
{
  
  __device__  bool operator()(const POS &a, const POS &b)
  {
    return (b.min_dist +b.is_in * FLT_MAX > a.min_dist + a.is_in * FLT_MAX);
    
    //return thrust::greater<double>()(b.min_dist +b.is_in * FLT_MAX,a.min_dist + a.is_in * FLT_MAX);
  }
};

void prim(thrust::host_vector<POS> &villesPosVecHost,const int nbVilles, double &distance_total)
{
  // copy to GPU memory
  cout << "start to copy GPU memory"<<endl;
  thrust::device_vector<POS> villesPosVecDev(nbVilles);
  cout <<"checkpoint1"<<endl;

  thrust::copy(villesPosVecHost.begin(), villesPosVecHost.end(), villesPosVecDev.begin());
  cout <<"checkpoint2"<<endl;
  thrust::host_vector<int> indexSequenceHost(nbVilles);
  thrust::sequence(indexSequenceHost.begin(), indexSequenceHost.end());
    cout <<"checkpoint4"<<endl;

  thrust::device_vector<int> indexSequenceDev(nbVilles);
  thrust::copy(indexSequenceHost.begin(), indexSequenceHost.end(), indexSequenceDev.begin());

  cout << "GPU memory copied" <<endl;
	// init Prime 
  /* 
	for(i = 1; i < nbVilles; i++)
	{
		min_dist[i] = distance[0][i];
		parent[i] = 0;
	}
  */
  thrust::for_each(indexSequenceDev.begin(), indexSequenceDev.end(), compute_min_dist_functor(villesPosVecDev.begin(), 0));
  thrust::for_each(villesPosVecDev.begin(), villesPosVecDev.end(), set_all_parent_functor(0));
  POS temp_pos = villesPosVecDev[0];
  temp_pos.is_in = true;
  villesPosVecDev[0] = temp_pos;


	//iteration of Prime
	
  int k;
  distance_total = 0;
	for(k = 0; k < nbVilles -1 ; k++)
	{
    /*
		// find the minimal min_dist outstide of S
		min_min_dist = FLT_MAX;

		for(i = 0; i < nbVilles; i++)
		{
      __assume_aligned(min_dist, VEC_ALIGN);
				if(min_dist[i] >0 && min_min_dist > min_dist[i])
				{
					min_min_dist = min_dist[i];
					min_min_dist_index = i;
				}
		}
  */
    thrust::device_vector<POS>::iterator minPosPtr = thrust::min_element(thrust::device,villesPosVecDev.begin(), villesPosVecDev.end(), compare_min_dist_functor());
    temp_pos = *minPosPtr;
    temp_pos.is_in = true;
    *minPosPtr = temp_pos;
    distance_total += temp_pos.min_dist;
    cout << "min_dist: "<< temp_pos.min_dist << " distance_total: "<< distance_total<<endl;
    //update the min_dist
    /*
    int dist_temp;

    float* distance_i_ptr =  distance[min_min_dist_index];
		for(j = 0; j < nbVilles; j++)
		{
      __assume_aligned(distance_i_ptr, VEC_ALIGN);
      dist_temp = distance_i_ptr[j];

			if(min_dist[j] > dist_temp)
			{
				min_dist[j] = dist_temp;
				parent[j] = min_min_dist_index;
			}
		}
    */
    thrust::for_each(indexSequenceDev.begin(), indexSequenceDev.end(), update_min_dist_functor(villesPosVecDev.begin(), temp_pos.index));
  }
   thrust::copy(villesPosVecDev.begin(), villesPosVecDev.end(), villesPosVecHost.begin());

}


#endif
